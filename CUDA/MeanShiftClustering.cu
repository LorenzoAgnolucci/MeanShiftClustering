#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : MeanShiftClustering.cu
 Author      : Lorenzo Agnolucci
 Version     :
 Copyright   : 
 Description : CUDA implementation of Mean Shift clustering algorithm
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <chrono>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "hip/hip_vector_types.h"
#include "Utils.h"

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

#define BANDWIDTH 2
#define NUM_DIMENSIONS 3
#define NUM_ITERATIONS 10
#define NUM_TESTS 15
#define TILE_WIDTH 64
#define BLOCK_DIM TILE_WIDTH

__global__ void NaiveMeanShift(float* shiftedPoints, const float* __restrict__ originalPoints, const unsigned numPoints) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float3 newPosition = make_float3(0.0, 0.0, 0.0);
	float totalWeight = 0.0;

	if(idx < numPoints){
		float x = shiftedPoints[idx];
		float y = shiftedPoints[idx + numPoints];
		float z = shiftedPoints[idx + 2 * numPoints];
		float3 shiftedPoint = make_float3(x, y, z);

		for(int i = 0; i < numPoints; i++){
			x = originalPoints[i];
			y = originalPoints[i + numPoints];
			z = originalPoints[i + 2 * numPoints];
			float3 originalPoint = make_float3(x, y, z);
			float3 difference = shiftedPoint - originalPoint;
			float squaredDistance = dot(difference, difference);
			float weight = std::exp((-squaredDistance) / (2 * powf(BANDWIDTH, 2)));
			newPosition += originalPoint * weight;
			totalWeight += weight;
		}
		newPosition /= totalWeight;
		shiftedPoints[idx] = newPosition.x;
		shiftedPoints[idx + numPoints] = newPosition.y;
		shiftedPoints[idx + 2 * numPoints] = newPosition.z;
	}
}

__global__ void TilingMeanShift(float* shiftedPoints, const float* __restrict__ originalPoints, const unsigned numPoints) {

	__shared__ float tile[TILE_WIDTH][3];

	int tx = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + tx;

	float3 newPosition = make_float3(0.0, 0.0, 0.0);
	float totalWeight = 0.0;

	for (int tile_i = 0; tile_i < (numPoints - 1) / TILE_WIDTH + 1; ++tile_i) {

		int tile_idx = tile_i * TILE_WIDTH + tx;

		if(tile_idx < numPoints){
			tile[tx][0] = originalPoints[tile_idx];
			tile[tx][1] = originalPoints[tile_idx + numPoints];
			tile[tx][2] = originalPoints[tile_idx + 2 * numPoints];
		}else{
			tile[tx][0] = 0.0;
			tile[tx][1] = 0.0;
			tile[tx][2] = 0.0;
		}

		__syncthreads();

		if(idx < numPoints){
			float x = shiftedPoints[idx];
			float y = shiftedPoints[idx + numPoints];
			float z = shiftedPoints[idx + 2 * numPoints];
			float3 shiftedPoint = make_float3(x, y, z);

			for(int i = 0; i < TILE_WIDTH; i++){
				if (tile[i][0] != 0.0 && tile[i][1] != 0.0 && tile[i][2] != 0.0) {
					float3 originalPoint = make_float3(tile[i][0], tile[i][1], tile[i][2]);
					float3 difference = shiftedPoint - originalPoint;
					float squaredDistance = dot(difference, difference);
					if(sqrt(squaredDistance) <= BANDWIDTH){
						float weight = std::exp((-squaredDistance) / (2 * powf(BANDWIDTH, 2)));
						newPosition += originalPoint * weight;
						totalWeight += weight;
					}
				}
			}
		}
		__syncthreads();
	}

	if(idx < numPoints){
		newPosition /= totalWeight;
		shiftedPoints[idx] = newPosition.x;
		shiftedPoints[idx + numPoints] = newPosition.y;
		shiftedPoints[idx + 2 * numPoints] = newPosition.z;
	}

}

int main(void)
{

	std::string fileName = "dataset/3D_data_100000.csv";
	std::vector<float> inputPoints = readPointsFromCSV(fileName);

	int numPoints = inputPoints.size() / NUM_DIMENSIONS;
	printf("Num points %d\n", numPoints);

	float totalElapsedTime = 0.0;

	thrust::device_vector<float> originalPoints = inputPoints;
	thrust::device_vector<float> shiftedPoints = inputPoints;

	for(int j = 0; j < NUM_TESTS; j++){
		originalPoints = inputPoints;
		shiftedPoints = inputPoints;

		float* originalPointer = thrust::raw_pointer_cast(&originalPoints[0]);
		float* shiftedPointer = thrust::raw_pointer_cast(&shiftedPoints[0]);

		dim3 gridDim = dim3(ceil((float) numPoints / BLOCK_DIM));
		dim3 blockDim = dim3(BLOCK_DIM);

		std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
		for(int i = 0; i < NUM_ITERATIONS; i++){
			TilingMeanShift<<<gridDim, blockDim>>> (shiftedPointer, originalPointer, numPoints);
			hipDeviceSynchronize();
		}

		std::chrono::high_resolution_clock::time_point end = std::chrono::high_resolution_clock::now();
		float elapsedTime = std::chrono::duration_cast<std::chrono::duration<float>>(end - start).count();
		totalElapsedTime += elapsedTime;
	}
	totalElapsedTime /= NUM_TESTS;
	printf("\nTiling Mean Shift elapsed time: %f\n", totalElapsedTime);


	totalElapsedTime = 0.0;

	for(int j = 0; j < NUM_TESTS; j++){
			originalPoints = inputPoints;
			shiftedPoints = inputPoints;

			float* originalPointer = thrust::raw_pointer_cast(&originalPoints[0]);
			float* shiftedPointer = thrust::raw_pointer_cast(&shiftedPoints[0]);

			dim3 gridDim = dim3(ceil((float) numPoints / BLOCK_DIM));
			dim3 blockDim = dim3(BLOCK_DIM);

			std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
			for(int i = 0; i < NUM_ITERATIONS; i++){
				NaiveMeanShift<<<gridDim, blockDim>>> (shiftedPointer, originalPointer, numPoints);
				hipDeviceSynchronize();
			}

			std::chrono::high_resolution_clock::time_point end = std::chrono::high_resolution_clock::now();
			float elapsedTime = std::chrono::duration_cast<std::chrono::duration<float>>(end - start).count();
			totalElapsedTime += elapsedTime;
	}
	totalElapsedTime /= NUM_TESTS;
	printf("\nNaive Mean Shift elapsed time: %f\n", totalElapsedTime);
/*

	thrust::host_vector<float> hostShiftedPoints = shiftedPoints;

	std::vector<float> outputPoints;
	std::vector<float> clusterPoints;



	outputPoints.resize(hostShiftedPoints.size());
	clusterPoints.resize(numPoints);
	thrust::copy(hostShiftedPoints.begin(), hostShiftedPoints.end(), outputPoints.begin());

	start = std::chrono::high_resolution_clock::now();

	std::vector<float3> clusters;

	float clusterEps = 5;

	for (int i = 0; i < numPoints; i++) {
		float x = hostShiftedPoints[i];
		float y = hostShiftedPoints[i + numPoints];
		float z = hostShiftedPoints[i + 2 * numPoints];
		float3 point = make_float3(x, y, z);
    	auto iter = clusters.begin();
        auto iterEnd = clusters.end();
        while(iter != iterEnd){
        	float3 difference = point - *iter;
        	float distance = sqrt(dot(difference, difference));
            if (distance <= clusterEps){
            	int clusterIndex =  iter - clusters.begin();
            	clusterPoints[i] = clusterIndex;
                break;
            }
            iter++;
        }
        if(iter == iterEnd){
            clusters.push_back(point);
            int clusterIndex = clusters.size() - 1;
            clusterPoints[i] = clusterIndex;
        }
    }

	end = std::chrono::high_resolution_clock::now();
	elapsedTime = std::chrono::duration_cast<std::chrono::duration<float>>(end - start).count();


	writeClustersToCSV("output.csv", inputPoints, clusterPoints, numPoints);

	printf("\nClustering elapsed time: %f", elapsedTime);
	printf("\nNum clusters: %lu", clusters.size());


*/
	return 0;
}
